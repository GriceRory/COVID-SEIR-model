#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""

#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <stdio.h>



#define beta constant[0]
#define epsilon constant[1]
#define alpha constant[2]
#define delta constant[3]
#define gamma constant[4]
#define testingRate constant[5]
#define population constant[6]
#define ICUbeds constant[7]

#define susceptible input[0]
#define exposed input[1]
#define presymptomatic input[2]
#define infectiousUntested input[3]
#define infectiousTested input[4]
#define recoveredUntested input[5]
#define recoveredTested input[6]
#define deaths input[7]



__global__ void rungeKutta4(double* input, double* constant, int steps, double stepSize, double** simulation) {
    //inputs, y0+h*k1/2, y0+h*k2/2, y0+h*k3, constants
    __shared__ double yValues[8 * 5];
    yValues[threadIdx.x] = input[threadIdx.x];
    yValues[threadIdx.x + 8 * 3] = constant[threadIdx.x];
    simulation[0][threadIdx.x] = input[threadIdx.x];

    double* Y0 = yValues;
    double* Y1 = &yValues[8];
    double* Y2 = &yValues[8 * 2];
    double* Y3 = &yValues[8 * 3];
    double* constants = &yValues[8 * 4];

        
    auto PartialDifferentialEquation = [] __device__(double* input, double* constant, int DifferentialEquation) {
        return input[DifferentialEquation];
        double IFR = 0.01;
        if (constant[6] * input[4] * 0.0125 > constant[7]) { 
            IFR = 0.02 - 0.01 * constant[7] / (constant[6] * input[4] * 0.0125);
        }
        switch (DifferentialEquation) {
        case 0:
            //  dSuseptible/dt
            return (-beta * susceptible * (epsilon * presymptomatic + infectiousTested + infectiousUntested));
        case 1:
            //  dExposed/dt
            return (beta * susceptible * (epsilon * presymptomatic + infectiousTested + infectiousUntested) - alpha * exposed);
        case 2:
            //  dPresymptomatic/dt
            return (alpha * exposed - delta * presymptomatic);
        case 3:
            //  dInfectedUntested/dt
            return (delta * presymptomatic - (gamma + testingRate) * infectiousUntested);
        case 4:
            //  dInfectedTested/dt
            return (testingRate * infectiousUntested - gamma * infectiousTested);
        case 5:
            //  dRecoveredUntested/dt
            return (gamma * infectiousUntested * (1 - IFR));
        case 6:
            //  dRecoveredTested/dt
            return (gamma * infectiousTested * (1 - IFR));
        case 7:
            //  Dead
            return (1 - susceptible - exposed - presymptomatic - infectiousTested - infectiousUntested - recoveredTested - recoveredUntested);
        }
        return 0.0;
    };
                            
    //this is an intellisense bug, the compiler handles it just fine.
    __syncthreads();

    double k1, k2, k3, k4;
    if (threadIdx.x != 0) { return; }
    for (int i = 1; i < steps; ++i) {
        for (int j = 0; j < 8; ++j) {

            k1 = stepSize * PartialDifferentialEquation(Y0, constants, j);
            Y1[threadIdx.x] = Y0[threadIdx.x] + k1 / 2;
            __syncthreads();

            k2 = stepSize * PartialDifferentialEquation(Y1, constants, j);
            Y2[threadIdx.x] = Y0[threadIdx.x] + k2 / 2;
            __syncthreads();

            k3 = stepSize * PartialDifferentialEquation(Y2, constants, j);
            Y3[threadIdx.x] = Y0[threadIdx.x] + k3;
            __syncthreads();

            k4 = stepSize * PartialDifferentialEquation(Y3, constants, j);
            __syncthreads();

            simulation[i][j] = simulation[i - 1][j] + (k1 + 2 * k2 + 2 * k3 + k4) / 6;
            Y0[j] = simulation[i][j];
            __syncthreads();
        }
    }
}

void printSimulationLayer(double* layer) {
    printf("susceptible, exposed, presymptomatic, infectiousUntested, infectiousTested, recoveredUntested, recoveredTested, Deaths\n");
    //printf("beta, epsilon, alpha, delta, gamma, testingRate, population, ICUbeds\n");
    for (int i = 0; i < 7; ++i) {
        printf("%f, ", layer[i]);
    }
    printf("%f\n\n", layer[7]);
}

void initializeConstants(double *constant, double *input) {
    alpha = 0.25;
    delta = 1;
    gamma = 0.1;
    epsilon = 0.15;
    double R0 = 2.5;
    beta = R0 / (epsilon / delta + 1 / gamma);
    testingRate = 0.1;

    population = 5000000;
    susceptible = 10;
    exposed = 0;
    presymptomatic = 0;
    ICUbeds = 500/population;
    infectiousUntested = 0;
    infectiousTested = 0;
    recoveredUntested = 0;
    recoveredTested = 0;
    deaths = 0;
}

double** simulate(double* inputs, double* constants, int steps, double stepSize) {
    double** h_simulation;
    double** d_simulation;
    hipHostMalloc(&h_simulation, steps * sizeof(double*), 0U);
    hipHostMalloc(&d_simulation, steps * sizeof(double*));

    for (int i = 0; i < steps; ++i) {
        double* h_temp;
        double* d_temp;
        hipHostMalloc(&h_temp, 8 * sizeof(double), 0U);
        hipMalloc(&d_temp, 8 * sizeof(double));

        h_simulation[i] = h_temp;
        d_simulation[i] = d_temp;
    }
    
    rungeKutta4<<<1, 8>>>(inputs, constants, steps, stepSize, d_simulation);

    for (int i = 0; i < steps; ++i) { hipMemcpy(h_simulation[i], d_simulation[i], 8 * sizeof(double), hipMemcpyDeviceToHost); }
    return h_simulation;
}

int main() {
    double* h_constant;
    double* h_input;
    hipHostMalloc(&h_constant, 8 * sizeof(double));
    hipHostMalloc(&h_input, 8 * sizeof(double));
    
    initializeConstants(h_constant, h_input);

    printSimulationLayer(h_constant);
    printSimulationLayer(h_input);
    printf("done initializing variables\n\n\n\n\n\n\n\n\n\n\n");


    double* d_constant;
    double* d_input;
    hipMalloc(&d_constant, 8 * sizeof(double));
    hipMalloc(&d_input, 8 * sizeof(double));
    hipMemcpy(d_constant, h_constant, 8 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_input, h_input, 8 * sizeof(double), hipMemcpyHostToDevice);

    int simulationSteps = 2 * 365;
    double simulationStepSize = 0.5;

    double** h_simulation = simulate(d_input, d_constant, simulationSteps, simulationStepSize);
    hipDeviceSynchronize();
    int error = hipGetLastError();

    for (int i = 0; i < simulationSteps; ++i) {
        printSimulationLayer(h_simulation[i]);
    }

    printf("no runtime errors!!! cuda error %d", error);
    return error;
}
